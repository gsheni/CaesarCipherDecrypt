#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

__global__ void convert2nums(char* line, char answer[]);

__global__ void convert2nums(char* line, char answer[]){
	int i;
	int current;
	char character;

	for( i = 0; line[i] != '\0'; i++) {
		current = (int) line[i];
		character = (char) (current -1);
		answer[i] = character;
	}
}

int main ( int argc, char *argv[] )
{
		if (argc != 2){
		printf ("Incorrect number of command line arugments.\r\n");
		exit(0);
	}

	FILE *file = fopen(argv[1], "r"); // "r" for read

	char* line = NULL;
	int lengthOfFile;

	if ( file != NULL ){

		fseek(file, 0, SEEK_END);
		lengthOfFile = ftell(file);

		line = (char *) malloc(lengthOfFile* sizeof(char) + 1 ) ;

		fseek (file , 0 , SEEK_SET);

		char current_line[lengthOfFile+1];
		while (fgets(current_line, sizeof(current_line), file)) {
	        line = current_line;
	    }
		fclose ( file );
	}
	else{
		printf ("File could not be opened.\r\n");
		printf ("File may not exist or incorrectly named.\r\n");
   }

   	char answer[lengthOfFile];

	char *dev_line;
	char *dev_answer;

	int size = lengthOfFile * sizeof(char);

	hipMalloc((void**)&dev_line, size);
	hipMalloc((void**)&dev_answer, size);

	hipMemcpy(dev_line, line, size, hipMemcpyHostToDevice);

	convert2nums<<<1, 1>>>(dev_line,dev_answer);

	hipDeviceSynchronize();

	hipMemcpy(answer, dev_answer, size, hipMemcpyDeviceToHost);
	
    hipFree(dev_answer);

	printf("Decoded Message is: \r\n%s",answer);

	exit (0);
}






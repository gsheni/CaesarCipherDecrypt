#include "hip/hip_runtime.h"
// Gaurav Sheni
// CSC 391 
// September 16, 2015
// Project 1


#include <stdio.h>
#include <string.h>
#include <stdlib.h>

//declaring kernel call
__global__ void decrement(char* line, char* answer);

__global__ void decrement(char *current, char* answer){
	int i = threadIdx.x;
	answer[i] = (char)( (int) current[i] - 1 );
}

int main ( int argc, char *argv[] )
{
	//check to make sure that there is only two command line arguments
	if (argc != 2){
		printf ("Incorrect number of command line arugments.\r\n");
		//exit with 1 because exit(1) indicates that there were an error
		exit(1);
	}

	//open the file to be read, give the first, argument
	//"r" is for read
	FILE *file = fopen(argv[1], "r"); 

	//line keeps the input data, inialize to null. 
	char* line = NULL;
	//for determining how many characters are in the input data
	int lengthOfFile;

	//make sure the file exists and is valid
	if ( file != NULL ){

		//first go to the end of the file and find out how many characters were counted
		fseek(file, 0, SEEK_END);
		//store number of charactes found.
		lengthOfFile = ftell(file);

		//allocate a character arry based on how many characters were found
		//we need to + 1 because we need room for the null terminator character
		line = (char *) malloc(lengthOfFile* sizeof(char) + 1 ) ;

		//go back to the beginning of the input file
		fseek (file , 0 , SEEK_SET);

		//
		char current_line[lengthOfFile+1];
		while (fgets(current_line, sizeof(current_line), file)) {
	        line = current_line;
	    }
		fclose ( file );
	}
	//error message if file does not exist. 
	else{
		printf ("File could not be opened.\r\n");
		printf ("File may not exist or the command line arugment is incorrectly named.\r\n");
		//exit with 1 because exit(1) indicates that there were an error
		exit(1);
   }
   	// printf ("Character Count = %d.\r\n", lengthOfFile);
   	char answer[lengthOfFile];

	char *dev_line;
	char *dev_answer;

	int size = lengthOfFile * sizeof(char);

	hipMalloc((void**)&dev_line, size);
	hipMalloc((void**)&dev_answer, size);

	hipMemcpy(dev_line, line, size, hipMemcpyHostToDevice);

	decrement<<< 1, lengthOfFile >>>(dev_line, dev_answer);

	hipDeviceSynchronize();

	hipMemcpy(answer, dev_answer, size, hipMemcpyDeviceToHost);
	
    hipFree(dev_line);
    hipFree(dev_answer);

	printf("Decoded --- Message is: \r\n%s",answer);

	exit (0);
}





